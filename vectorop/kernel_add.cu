//
// Created by mitom on 12/1/21.
//
#include <iostream>
#include <cstdlib>
#include <sys/time.h>
#include <hip/hip_runtime.h>

using namespace std;

__global__
void vecAddKernel(float* A_d, float* B_d, float* C_d, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) C_d[i] = A_d[i] + B_d[i];
}

int main() {

    int n = 10000000;
    cout << n << endl;

    // 定义分配内存大小
    size_t size = n * sizeof(float);

    // host memery
    float *a = (float *)malloc(size);
    float *b = (float *)malloc(size);
    float *c = (float *)malloc(size);

    for (int i = 0; i < n; i++) {
        float af = rand() / double(RAND_MAX);
        float bf = rand() / double(RAND_MAX);
        a[i] = af;
        b[i] = bf;
    }

    float *da = NULL;
    float *db = NULL;
    float *dc = NULL;

    // device memery
    hipMalloc((void **)&da, size);
    hipMalloc((void **)&db, size);
    hipMalloc((void **)&dc, size);

    // host->device拷贝
    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    // 计时结构体
    struct timeval t1, t2;

    int threadPerBlock = 256;
    int blockPerGrid = (n + threadPerBlock - 1)/threadPerBlock;  // 类似ceil()，节省内存.
    printf("threadPerBlock: %d \nblockPerGrid: %d \n",threadPerBlock,blockPerGrid);

    gettimeofday(&t1, NULL);

    vecAddKernel <<< blockPerGrid, threadPerBlock >>> (da, db, dc, n);

    gettimeofday(&t2, NULL);

    // device->host结果拷贝
    hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);

    double timeuse = (t2.tv_sec - t1.tv_sec) + (double)(t2.tv_usec - t1.tv_usec)/1000000.0;
    cout << timeuse << endl;

    // 释放内存
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(a);
    free(b);
    free(c);
    return 0;
}
