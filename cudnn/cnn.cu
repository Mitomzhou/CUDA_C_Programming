#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>
#include <cstdlib>
#include <vector>
#include <stdio.h>
#include <hipDNN.h>

__global__ void dev_const(float *px, float k) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    px[tid] = k;
}

__global__ void dev_iota(float *px) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    px[tid] = tid;
}

/**
 * 输出张量
 * @param data
 * @param n
 * @param c
 * @param h
 * @param w
 */
void print(const float *data, int n, int c, int h, int w) {
    std::vector<float> buffer(1 << 20);
    hipMemcpy(buffer.data(), data,n * c * h * w * sizeof(float), hipMemcpyDeviceToHost);
    int a = 0;
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < c; ++j) {
            std::cout << "n=" << i << ", c=" << j << ":" << std::endl;
            for (int k = 0; k < h; ++k) {
                for (int l = 0; l < w; ++l) {
                    std::cout << std::setw(4) << std::right << buffer[a];
                    ++a;
                }
                std::cout << std::endl;
            }
        }
    }
    std::cout << std::endl;
}


int main() {

    // 句柄
    hipdnnHandle_t cudnn_handle;
    hipdnnCreate(&cudnn_handle);

    // Input张量
    const int in_n = 1;
    const int in_c = 1;
    const int in_h = 5;
    const int in_w = 5;
    printf("Input:NCHW-(%d,%d,%d,%d)\n", in_n,in_c,in_h,in_w);

    // Input描述
    hipdnnTensorDescriptor_t in_desc;
    hipdnnCreateTensorDescriptor(&in_desc);
    hipdnnSetTensor4dDescriptor(in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, in_n, in_c, in_h, in_w);

    // Input分配内存
    float *in_data;
    hipMalloc(&in_data, in_n * in_c * in_h * in_w * sizeof(float));

    // Filter
    const int filt_k = 1;
    const int filt_c = 1;
    const int filt_h = 2;
    const int filt_w = 2;
    printf("Filter:NCHW-(%d,%d,%d,%d)\n", filt_k,filt_c,filt_h,filt_w);

    hipdnnFilterDescriptor_t filt_desc;
    hipdnnCreateFilterDescriptor(&filt_desc);
    hipdnnSetFilter4dDescriptor(filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, filt_k, filt_c, filt_h, filt_w);

    float *filt_data;
    hipMalloc(&filt_data, filt_k * filt_c * filt_h * filt_w * sizeof(float));

    // Convolution
    const int pad_h = 1;
    const int pad_w = 1;
    const int str_h = 1;
    const int str_w = 1;
    const int dil_h = 1;
    const int dil_w = 1;
    printf("Convolution:pad_h, pad_w, str_h, str_w, dil_h, dil_w: %d,%d,%d,%d,%d,%d \n", pad_h,pad_w,str_h,str_w,dil_h,dil_w);


    hipdnnConvolutionDescriptor_t conv_desc;
    hipdnnCreateConvolutionDescriptor(&conv_desc);
    hipdnnSetConvolution2dDescriptor(conv_desc,pad_h, pad_w, str_h, str_w, dil_h, dil_w,HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT);

    // Output
    int out_n;
    int out_c;
    int out_h;
    int out_w;

    hipdnnGetConvolution2dForwardOutputDim(conv_desc, in_desc, filt_desc, &out_n, &out_c, &out_h, &out_w);
    printf("Output:NCHW-(%d,%d,%d,%d)\n", out_n,out_c,out_h,out_w);
    std::cout << std::endl;
    hipdnnTensorDescriptor_t out_desc;
    hipdnnCreateTensorDescriptor(&out_desc);
    hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, out_n, out_c, out_h, out_w);

    float *out_data;
    hipMalloc(&out_data, out_n * out_c * out_h * out_w * sizeof(float));


    // Algorithm
    hipdnnConvolutionFwdAlgo_t algo;
    hipdnnGetConvolutionForwardAlgorithm(cudnn_handle,in_desc, filt_desc, conv_desc, out_desc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo);

    std::cout << "Convolution algorithm: " << algo << std::endl;
    std::cout << std::endl;

    // Workspace
    size_t ws_size;
    hipdnnGetConvolutionForwardWorkspaceSize(cudnn_handle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size);

    float *ws_data;
    hipMalloc(&ws_data, ws_size);

    std::cout << "Workspace size: " << ws_size << std::endl;
    std::cout << std::endl;

    // Perform
    float alpha = 1.f;
    float beta = 0.f;
    dev_iota<<<in_w * in_h, in_n * in_c>>>(in_data);
    dev_const<<<filt_w * filt_h, filt_k * filt_c>>>(filt_data, 1.f);
    hipdnnConvolutionForward(cudnn_handle, &alpha, in_desc, in_data, filt_desc, filt_data,conv_desc, algo, ws_data, ws_size, &beta, out_desc, out_data);

    // Results
    std::cout << "in_data:" << std::endl;
    print(in_data, in_n, in_c, in_h, in_w);

    std::cout << "filt_data:" << std::endl;
    print(filt_data, filt_k, filt_c, filt_h, filt_w);

    std::cout << "out_data:" << std::endl;
    print(out_data, out_n, out_c, out_h, out_w);

    // Finalizing
    hipFree(ws_data);
    hipFree(out_data);
    hipdnnDestroyTensorDescriptor(out_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipFree(filt_data);
    hipdnnDestroyFilterDescriptor(filt_desc);
    hipFree(in_data);
    hipdnnDestroyTensorDescriptor(in_desc);
    hipdnnDestroy(cudnn_handle);
    return 0;
}